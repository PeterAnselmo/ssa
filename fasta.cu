#include "hip/hip_runtime.h"
#ifndef FASTA_H
#define FASTA_H

#include <fstream>
#include <string>
#include <string.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "read.cu"

using namespace std;

class Fasta{
private:
    string _filename;
    string _description;
    string _seq;

public:
    Fasta(string new_filename){
        _filename = new_filename;
    }

    void description(string new_description){
        _description = new_description;
    }
    void seq(string new_seq){
        _seq = new_seq;
    }

    void write(int line_width = 80){
        ofstream fh;
        fh.open(_filename.c_str());
        if( fh.fail() ){
            cout << "Error opening Fasta file for writing.\n";
            exit(1);
        }
        fh << ">" << _description << endl;

        int num_lines = ceil(_seq.size()/80.0);
        for(int i=0; i<num_lines; ++i){
            fh << _seq.substr(i*line_width, line_width) << endl;
        }
    }

};

#endif
