#include "hip/hip_runtime.h"
#ifndef ASSEMBLY_H
#define ASSEMBLY_H

#include <algorithm>
#include <iomanip>
#include <list>
#include "contig.cu"
#include "fastqfile.cu"
#include "read.cu"
#include "sw_matrix.cpp"

using namespace std;

const bool DEBUGGING = true;

//number of bases in common with edges of consensus & read
//during perfect read assembly stage
const int MIN_OVERLAP = 20;

//max number of initial perfect match contigs to assemble
const unsigned int CONTIG_CAP = 1000;

//when trimming contigs, bases with lower than or eqaual to this quality
//will be removed from edges. Contigs consisting of only bases below
//this quality will be ommitted from contig assembly
const unsigned int CONTIG_TRIM_QUALITY = 1;

//mininum Conitig SW score to consider a match,
//this will need to be adjusted after adjusting match/mismatch scores
const int CONTIG_MATCH_THRESHOLD = 30;

class Assembly {
public:
    vector<Read> reads;
    list<Contig> contigs;

public:
    Assembly(FastqFile &input_file){
        reads = input_file.reads();
        if(reads.empty()){
            cout << "[ERROR] no reads to align." << endl;
            exit(1);
        }
    }

    //phase 1 - assemble contigs based on perfect overlap in reads - no mismatch allowed
    void assemble_perfect_contigs(){
        unsigned int pass = 0;
        //We'll create CONTIG_CAP constigs consisting of perfect matches between reads
        while(pass < CONTIG_CAP){
            Contig c(pass++);

            //find the first unmapped read with no "n" bases to seed the contig
            bool all_mapped = true;

            vector<Read>::iterator read;
            for(read = reads.begin(); read != reads.end(); ++read){
                if( !read->assembled()  && read->seq().find('N') == string::npos){
                    read->assemble(c.id(), 0);
                    c.seq(read->seq());
                    all_mapped = false;
                    break;
                }
            }

            //if all the reads were mapped before hitting the contig cap, exit
            if( all_mapped ){
                return;
            }

            if(DEBUGGING) {
                cout << "Starting new contig with sequence:\n" << c.seq() << endl;
            }

            //if any reads were mapped in the last iteration
            bool mapped_read = true;
            while(mapped_read){
                mapped_read = false;

                if(DEBUGGING) {
                    cout << "Restarting at the beginning of read list\n";
                }

                //consider all reads for contig assembly, take the first that matches with > MIN_OVERLAP
                vector<Read>::iterator read;
                for(read = reads.begin(); read != reads.end(); ++read){

                    //if this read was already mapped
                    if( read->assembled() ){
                        continue;
                    }

                    cout << "Considering read: " << read->seq() << endl;

                    unsigned int end_pos = c.size() - MIN_OVERLAP;

                    //compare right side of contig to left side of read
                    for(unsigned int i=0; i<end_pos; ++i){
                        
                        unsigned int compare_size = min(read->size(), c.size()-i);

                        if(DEBUGGING2){
                            cout << "Considering overlap: " << c.substr(i,compare_size) << "|" << read->substr(0,compare_size) << endl;
                        }
                        if( c.substr(i,compare_size) == read->substr(0,compare_size) ){
                            assemble_perfect_read(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                        if( c.substr(i,compare_size) == read->rev_comp().substr(0,compare_size) ){
                            read->set_rev_comp();
                            assemble_perfect_read(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                    }
                    
                    //compare left side of contig to right side of red
                    end_pos = read->size() - MIN_OVERLAP;
                    for(unsigned int i=0; i<end_pos; ++i){
                        
                        unsigned int compare_size = min(c.size(), read->size()-i);

                        if(DEBUGGING2){
                            cout << "Considering overlap: " << read->substr(i,compare_size) << "|" << c.substr(0,compare_size) << endl;
                        }
                        if( read->substr(i,compare_size) == c.substr(0,compare_size) ){
                            assemble_perfect_read_left(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                        if( c.substr(i,compare_size) == read->rev_comp().substr(0,compare_size) ){
                            read->set_rev_comp();
                            assemble_perfect_read_left(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                    }
                }
            }
            contigs.push_back(c);
        }
    }

    void assemble_perfect_contigs_cuda(){
        unsigned int pass = 0;
        //We'll create CONTIG_CAP constigs consisting of perfect matches between reads
        while(pass < CONTIG_CAP){
            Contig c(pass++);

            //find the first unmapped read with no "n" bases to seed the contig
            bool all_mapped = true;
            vector<Read>::iterator read;
            for(read = reads.begin(); read != reads.end(); ++read){
                if( !read->assembled()  && read->seq().find('N') == string::npos){
                    read->assemble(c.id(), 0);
                    c.seq(read->seq());
                    all_mapped = false;
                    break;
                }
            }

            //if all the reads were mapped before hitting the contig cap, exit
            if( all_mapped ){
                return;
            }

            if(DEBUGGING) {
                cout << "Starting new contig with sequence:\n" << c.seq() << endl;
            }

            //if any reads were mapped in the last iteration
            bool mapped_read = true;
            while(mapped_read){
                mapped_read = false;

                if(DEBUGGING) {
                    cout << "Restarting at the beginning of read list\n";
                }

                //consider all reads for contig assembly, take the first that matches with > MIN_OVERLAP
                vector<Read>::iterator read;
                for(read = reads.begin(); read != reads.end(); ++read){

                    //if this read was already mapped
                    if( read->assembled() ){
                        continue;
                    }

                    cout << "Considering read: " << read->seq() << endl;

                    unsigned int end_pos = c.size() - MIN_OVERLAP;

                    //compare right side of contig to left side of read
                    for(unsigned int i=0; i<end_pos; ++i){
                        
                        unsigned int compare_size = min(read->size(), c.size()-i);

                        if(DEBUGGING2){
                            cout << "Considering overlap: " << c.substr(i,compare_size) << "|" << read->substr(0,compare_size) << endl;
                        }
                        if( c.substr(i,compare_size) == read->substr(0,compare_size) ){
                            assemble_perfect_read(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                        if( c.substr(i,compare_size) == read->rev_comp().substr(0,compare_size) ){
                            read->set_rev_comp();
                            assemble_perfect_read(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                    }
                    
                    //compare left side of contig to right side of red
                    end_pos = read->size() - MIN_OVERLAP;
                    for(unsigned int i=0; i<end_pos; ++i){
                        
                        unsigned int compare_size = min(c.size(), read->size()-i);

                        if(DEBUGGING2){
                            cout << "Considering overlap: " << read->substr(i,compare_size) << "|" << c.substr(0,compare_size) << endl;
                        }
                        if( read->substr(i,compare_size) == c.substr(0,compare_size) ){
                            assemble_perfect_read_left(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                        if( c.substr(i,compare_size) == read->rev_comp().substr(0,compare_size) ){
                            read->set_rev_comp();
                            assemble_perfect_read_left(c, *read, i);
                            mapped_read = true;
                            break;
                        }
                    }
                }
            }
            contigs.push_back(c);
        }

    }

    //phase 2 - assemble contigs to eeach other, allowing mismatches
    void assemble_contigs(){

        //loop over all the contigs, compare them to each other
        list<Contig>::iterator c1;
        for(c1 = contigs.begin(); c1 != contigs.end(); ++c1){

            list<Contig>::iterator c2;
            for(c2 = contigs.begin(); c2 != contigs.end(); ++c2){
                if(c1->id() == c2->id()){
                    continue;
                }
                SWMatrix m(c1->seq(), c2->seq());
                if(DEBUGGING3){
                    m.print_matrix();
                }
                if(m.score() > CONTIG_MATCH_THRESHOLD){
                    m.gap_seqs();
                    c1->merge(*c2);
                }
            }


            /*
            //if any reads were mapped in the last iteration
            bool mapped_read = true;
            while(mapped_read){
            mapped_read = false;

                if(DEBUGGING) {
                    cout << "Restarting at the beginning of read list\n";
                }

                //loop over all unmatched reads
                vector<Read>::iterator iter;
                for( iter = reads.begin(); iter != reads.end(); ++iter){
                    Read read = *iter;

                    //if this read was already mapped
                    if( read.assembled() ){
                        continue;
                    }

                    int high_score = 0;
                    int high_pos = 0;
                    vector<Read>::iterator high_iter;

                    unsigned int end_pos = c.size() - MIN_OVERLAP;
                    for(unsigned int i=0; i<end_pos; ++i){
                        unsigned int compare_size = min({read.size(), c.size()-i});

                        SWMatrix sw(c.substr(i,compare_size), read.substr(0,compare_size));
                        SWMatrix rev_sw(c.substr(i,compare_size), read.rev_comp().substr(0,compare_size));

                        if(DEBUGGING2){
                            cout << "Score at pos: " << i << ":" << sw.score() << endl;
                            cout << "RC Score at pos: " << i << ":" << rev_sw.score() << endl;
                        }
                        if( sw.score() > MATCH_THRESHOLD && sw.score() > high_score ){
                            high_score = sw.score();
                            high_iter = iter;
                            high_pos = i;
                        }
                        if( rev_sw.score() > MATCH_THRESHOLD && rev_sw.score() > high_score ){
                            high_score = rev_sw.score();
                            high_iter = iter;
                            high_pos = i;
                            read.set_rev_comp();
                        }
                    }
                    //ignoring assemblies at position 0, likely means that right half of seq
                    //aligned to before start of reference
                    if( high_score != 0 && high_pos != 0 ){
                        assemble_read(c, *high_iter, high_pos);
                        mapped_read = true;
                    }
                }
            }
            */
        }
    }

    void trim_contigs(){
        list<Contig>::iterator iter = contigs.begin();
        while(iter != contigs.end()){
            iter->trim(CONTIG_TRIM_QUALITY, reads);
            if(iter->size() == 0){
                if(DEBUGGING){
                    cout << "Deleting low-quality contig: " << iter->id() << endl;
                }
                vector<Read>::iterator read;
                for(read = reads.begin(); read != reads.end(); ++read){
                    if( read->assembled() && read->assem_contig == iter->id() ){
                        read->unassemble();
                    }
                }
                contigs.erase(iter++);
            } else {
                ++iter;
            }
        }

    }

    void print_report(){
        unsigned int num_assembled = 0;
        for(vector<Read>::iterator read = reads.begin(); read != reads.end(); ++read){
            if( read->assembled() ){
                ++num_assembled;
            }
        }
        cout << "Assembled " << num_assembled << "/" << reads.size() << " reads ("
             << static_cast<double>(num_assembled) * 100 /reads.size() << "%)" << endl;

    }

    void print_contigs(bool show_reads = true){

        //sort the reads before displaying below contigs
        //reads.sort([](const Read &r1, const Read &r2){ return r1.assem_pos < r2.assem_pos; });

        for(list<Contig>::iterator c = contigs.begin(); c != contigs.end(); ++c){
            cout << "Assembled Contig " << c->id() << ":\n" << c->seq() << endl;
            cout << c->qual() << endl;
            if(show_reads){
                for(vector<Read>::iterator read = reads.begin(); read != reads.end(); ++read){
                    if(read->assembled() && read->assem_contig == c->id()){
                        for(int i=0; i<read->assem_pos; ++i){
                            cout << " ";
                        }
                        cout << read->gapped_seq << endl;
                    }
                }
            }
        }
    }

private:
    void assemble_read(Contig &c, Read &read, unsigned int pos){

        unsigned int overlap_size = min(read.size(), c.size()-pos);

        if(DEBUGGING){
            cout << "Assembling Read: " << read.seq() << " to contig " << c.id() << " at " << pos << endl;
        }
        read.assemble(c.id(), pos);

        SWMatrix sw(c.substr(pos, read.size()), read.seq());
        sw.gap_seqs();
        read.gapped_seq = sw.get_gapped_seq2();

        //if the read extends to the right of the c.seq
        if( overlap_size < read.gapped_seq.size() ){
            if(DEBUGGING){
                cout << "overlap size: " << overlap_size << endl;
                cout << "Read size: " << read.seq().size() << endl;
            }
            for(unsigned int i=0; i<overlap_size; ++i){
                if(c.seq()[pos+i] == read.gapped_seq[i] ){
                    c.inc_qual(pos+i);
                }
            }
            string new_seq = read.gapped_seq.substr(overlap_size);
            if(DEBUGGING){
                cout << "adding " << new_seq << " to reference.\n";
            }
            c.append(new_seq);
        }
    }

    void assemble_perfect_read(Contig &c, Read &read, unsigned int pos){
        unsigned int overlap_size = min(read.size(), c.size()-pos);

        if(DEBUGGING){
            cout << "Assembling Read: " << read.seq() << " to contig " << c.id() << " at " << pos << " "
                 << "overlap size: " << overlap_size << endl;
        }

        //assemble the read here
        read.assemble(c.id(), pos);

        //increment the quality for all of the overlapping bases at the right of the contig
        for(unsigned int j=0; j<overlap_size; ++j){
            c.inc_qual(pos+j);
        }

        if( overlap_size < read.size() ){
            string new_seq = read.seq().substr(overlap_size);

            if(DEBUGGING){
                cout << "adding " << new_seq << " to reference, overlap size: " << overlap_size << ".\n";
            }
            c.append(new_seq);

            if(DEBUGGING){
                cout << "New Reference:\n" << c.seq() << endl;
            }
        }
    }

    void assemble_perfect_read_left(Contig &c, Read &read, unsigned int read_pos){
        unsigned int overlap_size = min(c.size(), read.size()-read_pos);

        if(DEBUGGING){
            cout << "Assembling Read: " << read.seq() << " to contig " << c.id() << " at read pos " << read_pos << " "
                 << "overlap size: " << overlap_size << endl;
        }

        //increment the quality for all of the overlapping bases at the left of the contig
        for(unsigned int j=0; j<overlap_size; ++j){
            c.inc_qual(j);
        }

        if( overlap_size < read.size() ){
            string new_seq = read.seq().substr(0,read_pos);

            if(DEBUGGING){
                cout << "adding " << new_seq << " to beginning reference, overlap size: " << overlap_size << ".\n";
            }
            c.prepend(new_seq);
            c.unshift_aligned_reads(new_seq.size(), reads);

            if(DEBUGGING){
                cout << "New Reference:\n" << c.seq() << endl;
            }
        }
       
        //assemble the read here
        //do it after prepending to sequence, so it is not shifted
        read.assemble(c.id(), 0);
    }

};

#endif
