#include "hip/hip_runtime.h"
#ifndef ASSEMBLY_H
#define ASSEMBLY_H

#include <algorithm>
#include <iomanip>
#include <list>
#include "stdio.h"
#include "contig.cu"
#include "fastqfile.cu"
#include "read.cu"
#include "sw_matrix.cu"

using namespace std;

__host__ __device__ void assemble_perfect_read(Contig *c, Read &read, unsigned int pos);
__host__ __device__ void assemble_perfect_read_left(Contig *c, Read &read, unsigned int pos);
__global__ void map_reads_on_device(Contig *c, Read *reads, int num_reads);

class Assembly {
public:
    Read* reads;
    int num_reads;
    list<Contig> contigs;

public:
    Assembly(FastqFile &input_file){
        reads = input_file.reads();
        num_reads = input_file.num_reads();
        if(num_reads == 0){
            cout << "[ERROR] no reads to align." << endl;
            exit(1);
        }
    }

    //phase 1 - assemble contigs based on perfect overlap in reads - no mismatch allowed
    void assemble_perfect_contigs(){
        unsigned int pass = 0;
        //We'll create CONTIG_CAP constigs consisting of perfect matches between reads
        while(pass < CONTIG_CAP){
            Contig c(pass++);
            Contig *cPtr = &c;

            //find the first unmapped read with no "n" bases to seed the contig
            bool all_mapped = true;

            for(int i=0; i<num_reads; ++i){
                if( !reads[i].assembled() && reads[i].find('N') == reads[i].size()){
                    reads[i].assemble(c.id(), 0);
                    c.set_seq(reads[i].seq());
                    all_mapped = false;
                    break;
                }
            }

            //if all the reads were mapped before hitting the contig cap, exit
            if( all_mapped ){
                return;
            }

            if(DEBUGGING) {
                cout << "Starting new contig with sequence:\n" << c.seq() << endl;
            }

            //if any reads were mapped in the last iteration
            bool mapped_read = true;
            while(mapped_read){
                mapped_read = false;

                if(DEBUGGING) {
                    cout << "Restarting at the beginning of read list\n";
                }

                //consider all reads for contig assembly, take the first that matches with > MIN_OVERLAP
                for(int i=0; i<num_reads; ++i){

                    //if this read was already mapped
                    if( reads[i].assembled() ){
                        continue;
                    }

                    if(DEBUGGING2){
                        printf("Considering read: %s\n", reads[i].seq());
                    }

                    //unsigned int start_pos = c.size() - reads[i].size();
                    unsigned int end_pos = c.size() - MIN_OVERLAP;

                    //compare right side of contig to left side of read
                    //start position depends on whether trying to align reads
                    //for(unsigned int i=start_pos; i<end_pos; ++i){
                    for(unsigned int j=0; j<end_pos; ++j){
                        
                        unsigned int compare_size = min(reads[i].size(), c.size()-j);

                        char *read_substr = reads[i].substr(0,compare_size);
                        char *contig_substr = c.substr(j,compare_size);
                        if(DEBUGGING2){
                            printf("Considering overlap: %s | %s\n", contig_substr, read_substr);
                        }
                        if( !reads[i].assembled() && strcmp(contig_substr, read_substr) == 0){
                            assemble_perfect_read(cPtr, reads[i], j);
                            mapped_read = true;
                        }
                        free(read_substr);
                        char *read_rev_substr = reads[i].rev_substr(0,compare_size);
                        if(DEBUGGING2){
                            printf("Considering overlap: %s | %s\n", contig_substr, read_rev_substr);
                        }
                        if( !reads[i].assembled() && strcmp(contig_substr, read_rev_substr) == 0 ){
                            reads[i].set_rev_comp();
                            assemble_perfect_read(cPtr, reads[i], j);
                            mapped_read = true;
                        }
                        free(read_rev_substr);
                        free(contig_substr);
                    }
                    
                    //compare left side of contig to right side of read
                    end_pos = reads[i].size() - MIN_OVERLAP;
                    for(unsigned int j=0; j<end_pos; ++j){
                        
                        unsigned int compare_size = min(c.size(), reads[i].size()-j);

                        char *read_substr = reads[i].substr(j,compare_size);
                        char *contig_substr = c.substr(0,compare_size);
                        if(DEBUGGING2){
                            printf("Considering overlap: %s | %s\n", read_substr, contig_substr);
                        }
                        if( !reads[i].assembled() && strcmp(read_substr, contig_substr) == 0 ){
                            assemble_perfect_read_left(cPtr, reads[i], j);
                            mapped_read = true;
                        }
                        free(read_substr);
                        char *read_rev_substr = reads[i].rev_substr(0,compare_size);
                        if(DEBUGGING2){
                            printf("Considering overlap: %s | %s\n", read_rev_substr, contig_substr);
                        }
                        if( !reads[i].assembled() && strcmp(contig_substr, read_rev_substr) == 0 ){
                            reads[i].set_rev_comp();
                            assemble_perfect_read_left(cPtr, reads[i], j);
                            mapped_read = true;
                        }
                        free(read_rev_substr);
                        free(contig_substr);
                    }
                }
            }
            contigs.push_back(c);
        }
    }

    void assemble_perfect_contigs_cuda(){

        //copy all reads to device
        Read *d_reads;
        int reads_size = num_reads * sizeof(Read);
        hipMalloc( (void**)&d_reads, reads_size);
        hipMemcpy( d_reads, reads, reads_size, hipMemcpyHostToDevice);

        int threads_per_block = 256;
        int num_blocks = num_reads / threads_per_block + 1;

        unsigned int pass = 0;
        //We'll create CONTIG_CAP constigs consisting of perfect matches between reads
        while(pass < CONTIG_CAP){
            Contig c(pass++);
            Contig *cPtr = &c;

            //find the first unmapped read with no "n" bases to seed the contig
            bool all_mapped = true;

            for(int i=0; i<num_reads; ++i){
                if( !reads[i].assembled() && reads[i].find('N') == reads[i].size()){
                    reads[i].assemble(c.id(), 0);
                    c.set_seq(reads[i].seq());
                    all_mapped = false;
                    break;
                }
            }

            //if all the reads were mapped before hitting the contig cap, exit
            if( all_mapped ){
                return;
            }

            if(DEBUGGING) {
                cout << "Starting new contig with sequence:\n" << c.seq() << endl;
            }

            //move contig onto device
            Contig *d_contig;
            hipMalloc( (void**)&d_contig, sizeof(Contig));
            hipMemcpy( d_contig, &c, sizeof(Contig), hipMemcpyHostToDevice);

            //if any reads were mapped in the last iteration
            bool mapped_read = true;
            while(mapped_read){
                mapped_read = false;

                if(DEBUGGING) {
                    cout << "Restarting at the beginning of read list\n";
                }

                //consider all reads for contig assembly, take the first that matches with > MIN_OVERLAP
                map_reads_on_device<<<num_blocks, threads_per_block>>>(d_contig, d_reads, num_reads);

                hipDeviceSynchronize();
                hipMemcpy( cPtr, d_contig, sizeof(Contig), hipMemcpyDeviceToHost);

                contigs.push_back(*cPtr);
            }
        }

        //copy reads back from device
        hipMemcpy( reads, d_reads, reads_size, hipMemcpyDeviceToHost);
    }

    //phase 2 - assemble contigs to eeach other, allowing mismatches
    void assemble_contigs(){
        //keep looping as long as there a high quality matches
        bool had_merge = true;

        while(had_merge){
            if(DEBUGGING){
                printf("Restarting Contig Merge Loop.\n");
            }
            had_merge = false;
            //loop over all the contigs, compare them to each other
            list<Contig>::iterator c1;
            for(c1 = contigs.begin(); c1 != contigs.end(); ++c1){
                if(DEBUGGING){
                    printf("Comparing contig %d against others\n", c1->id());
                }
                //needs to be while rather then for to handle deleting elements real time
                list<Contig>::iterator c2 = contigs.begin();;
                while(c2 != contigs.end()){
                    if(c1->id() == c2->id()){
                        ++c2;
                        continue;
                    }

                    //compute the score matrix using the sequences from the two contigs
                    SWMatrix m(*c1, *c2);
                    Contig* rev_c2 = c2->rev_comp();
                    SWMatrix m2(*c1, *rev_c2);
                    if(DEBUGGING2){
                        printf("Matrix Score: %d\n", m.score());
                    }

                    //if these two contigs are a match, merge the second one
                    //into the first one and delete the second.
                    if(m.score() >= CONTIG_MATCH_THRESHOLD){
                        if(DEBUGGING){
                            printf("Merging Contigs %d & %d\n", c1->id(), c2->id());
                        }
                        had_merge = true;
                        m.merge_seqs();
                        c1->set_seq(m.complete_seq(), false);
                        c1->set_qual(m.complete_qual());
                        contigs.erase(c2++);

                    //if the rev_comp of the second matches, merge it into the first
                    } else if (m2.score() > CONTIG_MATCH_THRESHOLD ){
                        if(DEBUGGING){
                            printf("Merging Contigs %d & rev %d\n", c1->id(), c2->id());
                        }
                        had_merge = true;
                        m2.merge_seqs();
                        c1->set_seq(m2.complete_seq(), false);
                        c1->set_qual(m2.complete_qual());
                        contigs.erase(c2++);
                    } else {
                        ++c2;
                    }
                    delete(rev_c2);
                }//inner contigs
            }//outer contigs
        }//while had merge
    }

    char* final_seq(){
        list<Contig>::iterator max;
        unsigned int max_size = 0;

        for(list<Contig>::iterator c = contigs.begin(); c != contigs.end(); ++c){
            if(c->size() > max_size){
                max_size = c->size();
                max = c;
            }
        }
        if(max_size == 0){
            printf("Error, unable to select final contig");
        }
        return max->seq();
    }


    void trim_contigs(){
        list<Contig>::iterator contig = contigs.begin();
        while(contig != contigs.end()){
            contig->trim(CONTIG_TRIM_QUALITY, reads, num_reads);
            if(contig->size() == 0){
                if(DEBUGGING){
                    printf("Deleting low-quality contig: %d\n", contig->id());
                }
                contigs.erase(contig++);
            } else {
                ++contig;
            }
        }
    }

    void print_report(){
        unsigned int num_assembled = 0;
        for(int i=0; i<num_reads; ++i){
            if( reads[i].assembled() ){
                ++num_assembled;
            }
        }
        printf("Assembled %u/%u reads(%f%%)\n", num_assembled, 
                                              num_reads,
                                              static_cast<double>(num_assembled) * 100 /num_reads);

    }

    void print_contigs(bool show_reads = true){

        if(show_reads){
            //sort the reads before displaying below contigs
            //sort(reads.begin(), reads.end());
        }

        for(list<Contig>::iterator c = contigs.begin(); c != contigs.end(); ++c){
            printf("Assembled Contig %d:\n%s\n%s\n", c->id(), c->seq(), c->qual());
            if(show_reads){
                for(int i=0; i<num_reads; ++i){
                    if(reads[i].assembled() && reads[i].contig() == c->id()){
                        for(unsigned int j=0; j<reads[i].position(); ++j){
                            printf(" ");
                        }
                        printf("%s\n",reads[i].gapped_seq());
                    }
                }
            }
        }
    }

private:

};

    __host__ __device__ void assemble_perfect_read(Contig *c, Read &read, unsigned int pos){
        unsigned int overlap_size;
        if(read.size() < (c->size()-pos)){
           overlap_size = read.size();
        } else {
           overlap_size = c->size()-pos;
        }

        if(DEBUGGING){
            printf("Assembling read: %s to contig %d at %d, overlap size: %u\n", read.seq(), c->id(), pos, overlap_size);
        }

        //assemble the read here
        read.assemble(c->id(), pos);

        //increment the quality for all of the overlapping bases at the right of the contig
        for(unsigned int j=0; j<overlap_size; ++j){
            c->inc_qual(pos+j);
        }

        if( overlap_size < read.size() ){
            char *new_seq = read.substr(overlap_size);

            if(DEBUGGING){
                printf("Adding %s to end of reference\n", new_seq);
            }
            c->append(new_seq);
            free(new_seq);

            if(DEBUGGING){
                printf("New Reference:\n%s\n", c->seq());
            }
        }
    }

    __host__ __device__ void assemble_perfect_read_left(Contig *c, Read &read, unsigned int pos){
        unsigned int overlap_size;
        if(read.size() < (c->size()-pos)){
           overlap_size = read.size();
        } else {
           overlap_size = c->size()-pos;
        }


        if(DEBUGGING){
            printf("Assembling Read: %s to contig %d at pos %d with overlap %d\n", read.seq(), c->id(), pos, overlap_size);
        }

        //increment the quality for all of the overlapping bases at the left of the contig
        for(unsigned int j=0; j<overlap_size; ++j){
            c->inc_qual(j);
        }

        if( overlap_size < read.size() ){
            char *new_seq = read.substr(0,pos);

            if(DEBUGGING){
                printf("Adding %s to beginning of reference\n", new_seq);
            }
            c->prepend(new_seq);
            #ifndef __CUDA_ARCH__
//                c->unshift_aligned_reads(strlen(new_seq), reads, num_reads);
            #else
 //               c->unshift_aligned_reads(cudaStrlen(new_seq), reads, num_reads);
            #endif

            if(DEBUGGING){
                printf("New Reference:\n%s\n", c->seq());
            }
            free(new_seq);
        }
       
        //assemble the read here
        //do it after prepending to sequence, so it is not shifted
        read.assemble(c->id(), 0);
    }


    __global__ void map_reads_on_device(Contig *c, Read *reads, int num_reads){
        int tid = blockIdx.x*blockDim.x + threadIdx.x;

        //there may be a few extra threads called, make sure in range
        if( tid >= num_reads ){
            return;
        }

        //if this read was already mapped
        if( reads[tid].assembled() ){
            return;
        }

        if(DEBUGGING2){
            printf("Considering read: %s\n", reads[tid].seq());
        }

        //unsigned int start_pos = c.size() - reads[tid].size();
        unsigned int end_pos = c->size() - MIN_OVERLAP;

        //compare right side of contig to left side of read
        //start position depends on whether trying to align reads
        //for(unsigned int i=start_pos; i<end_pos; ++i){
        for(unsigned int j=0; j<end_pos; ++j){
            
            unsigned int compare_size = min(reads[tid].size(), c->size()-j);

            char *read_substr = reads[tid].substr(0,compare_size);
            char *contig_substr = c->substr(j,compare_size);
            if(DEBUGGING2){
                printf("Considering overlap: %s | %s\n", contig_substr, read_substr);
            }
            if( !reads[tid].assembled() && cudaStrcmp(contig_substr, read_substr) == 0){
                assemble_perfect_read(c, reads[tid], j);
            }
            free(read_substr);
            char *read_rev_substr = reads[tid].rev_substr(0,compare_size);
            if(DEBUGGING2){
                printf("Considering overlap: %s | %s\n", contig_substr, read_rev_substr);
            }
            if( !reads[tid].assembled() && cudaStrcmp(contig_substr, read_rev_substr) == 0 ){
                reads[tid].set_rev_comp();
                assemble_perfect_read(c, reads[tid], j);
            }
            free(read_rev_substr);
            free(contig_substr);
        }
        
        //compare left side of contig to right side of read
        end_pos = reads[tid].size() - MIN_OVERLAP;
        for(unsigned int j=0; j<end_pos; ++j){
            
            unsigned int compare_size = min(c->size(), reads[tid].size()-j);

            char *read_substr = reads[tid].substr(j,compare_size);
            char *contig_substr = c->substr(0,compare_size);
            if(DEBUGGING2){
                printf("Considering overlap: %s | %s\n", read_substr, contig_substr);
            }
            if( !reads[tid].assembled() && cudaStrcmp(read_substr, contig_substr) == 0 ){
                assemble_perfect_read_left(c, reads[tid], j);
            }
            free(read_substr);
            char *read_rev_substr = reads[tid].rev_substr(0,compare_size);
            if(DEBUGGING2){
                printf("Considering overlap: %s | %s\n", read_rev_substr, contig_substr);
            }
            if( !reads[tid].assembled() && cudaStrcmp(contig_substr, read_rev_substr) == 0 ){
                reads[tid].set_rev_comp();
                assemble_perfect_read_left(c, reads[tid], j);
            }
            free(read_rev_substr);
            free(contig_substr);
        }
    }


#endif
